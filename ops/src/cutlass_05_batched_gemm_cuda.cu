#include "hip/hip_runtime.h"

#include <iostream>
#include <vector>

#include "cutlass/cutlass.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/device/gemm_array.h"
#include "cutlass/gemm/device/gemm_batched.h"
#include "cutlass/numeric_types.h"

#pragma warning(disable : 4503)

// hipError_t cutlass_strided_batched_sgemm(
//     int m,
//     int n,
//     int k,
//     float alpha,
//     float const *A,
//     int lda,
//     long long int batch_stride_A,
//     float const *B,
//     int ldb,
//     long long int batch_stride_B,
//     float *C,
//     int ldc,
//     long long int batch_stride_C,
//     float beta,
//     int batch_count)
// {

//     using Gemm = cutlass::gemm::device::GemmBatched<
//         float, cutlass::layout::RowMajor,
//         float, cutlass::layout::RowMajor,
//         float, cutlass::layout::RowMajor>;

//     Gemm gemm_op;

//     cutlass::Status status = gemm_op({{m, n, k},
//                                       {A, lda},
//                                       batch_stride_A,
//                                       {B, ldb},
//                                       batch_stride_B,
//                                       {C, ldc},
//                                       batch_stride_C,
//                                       {C, ldc},
//                                       batch_stride_C,
//                                       {alpha, beta},
//                                       batch_count});

//     if (status != cutlass::Status::kSuccess)
//     {
//         return hipErrorUnknown;
//     }

//     return hipSuccess;
// }


// void launcher_batched_gemm_float(const int batch_count, const float* mat_A, const float* mat_B, float* mat_C, const int M, const int N, const int K, const float alpha, const float beta)
// {
//     // A, B are non-transpose, column major
//     int const lda = K;
//     int const ldb = N;
//     int const ldc = N;

//     // the memory is batched along M dimension for A, K dimension for B, and M dimension for C
//     long long int batch_stride_A = static_cast<long long int>(M) * static_cast<long long int>(lda);
//     long long int batch_stride_B = static_cast<long long int>(K) * static_cast<long long int>(ldb);
//     long long int batch_stride_C = static_cast<long long int>(M) * static_cast<long long int>(ldc);

//     cutlass_strided_batched_sgemm(
//         M, N, K, alpha,
//         mat_A, lda, batch_stride_A, mat_B, ldb, batch_stride_B, mat_C, ldc, batch_stride_C, 
//         beta, batch_count);
// }


hipError_t cutlass_strided_batched_hgemm(
    int m,
    int n,
    int k,
    cutlass::half_t alpha,
    cutlass::half_t const *A,
    int lda,
    long long int batch_stride_A,
    cutlass::half_t const *B,
    int ldb,
    long long int batch_stride_B,
    cutlass::half_t *C,
    int ldc,
    long long int batch_stride_C,
    cutlass::half_t beta,
    int batch_count)
{

    using Gemm = cutlass::gemm::device::GemmBatched<
        cutlass::half_t, cutlass::layout::RowMajor,
        cutlass::half_t, cutlass::layout::RowMajor,
        cutlass::half_t, cutlass::layout::RowMajor>;

    Gemm gemm_op;

    cutlass::Status status = gemm_op({{m, n, k},
                                      {A, lda},
                                      batch_stride_A,
                                      {B, ldb},
                                      batch_stride_B,
                                      {C, ldc},
                                      batch_stride_C,
                                      {C, ldc},
                                      batch_stride_C,
                                      {alpha, beta},
                                      batch_count});

    if (status != cutlass::Status::kSuccess)
    {
        return hipErrorUnknown;
    }

    return hipSuccess;
}


void launcher_batched_gemm_half(const int batch_count, const __half* mat_A, const __half* mat_B, __half* mat_C, int M, int N, int K, float alpha, float beta)
{
    // A, B are non-transpose, column major
    int const lda = K;
    int const ldb = N;
    int const ldc = N;

    cutlass::half_t alpha_cutlass = cutlass::half_t(alpha);
    cutlass::half_t beta_cutlass = cutlass::half_t(beta);

    const cutlass::half_t* mat_A_cutlass = reinterpret_cast<const cutlass::half_t*>(mat_A);
    const cutlass::half_t* mat_B_cutlass = reinterpret_cast<const cutlass::half_t*>(mat_B);
    cutlass::half_t* mat_C_cutlass = reinterpret_cast<cutlass::half_t*>(mat_C);

    // the memory is batched along M dimension for A, K dimension for B, and M dimension for C
    long long int batch_stride_A = static_cast<long long int>(M) * static_cast<long long int>(lda);
    long long int batch_stride_B = static_cast<long long int>(K) * static_cast<long long int>(ldb);
    long long int batch_stride_C = static_cast<long long int>(M) * static_cast<long long int>(ldc);

    cutlass_strided_batched_hgemm(
        M, N, K, alpha_cutlass,
        mat_A_cutlass, lda, batch_stride_A, 
        mat_B_cutlass, ldb, batch_stride_B, 
        mat_C_cutlass, ldc, batch_stride_C,
        beta_cutlass, batch_count);
}
