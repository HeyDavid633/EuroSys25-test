
#include <hip/hip_runtime.h>
#include <cstdio>

#define THREADS_PER_BLOCK 256
#define WARP_SIZE 32
#define DIVUP(m, n) ((m + n - 1) / n)


__global__ void tensoradd_kernel(const float* a, const float* b, float * c, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n){
        c[idx] = a[idx] + b[idx];
    }
}


void tensoradd_launcher(const float* a, const float* b, float* c, int n){
    dim3 blockSize(DIVUP(n, THREADS_PER_BLOCK));
    dim3 threadSize(THREADS_PER_BLOCK);
    tensoradd_kernel<<<blockSize, threadSize>>>(a, b, c, n);
}