// 10.14 cutlass: gemm

#include <iostream>
#include <sstream>
#include <vector>

#include "helper.h"
#include "cutlass/gemm/device/gemm.h"


hipError_t CutlassSgemmNN_float(
    int M,
    int N,
    int K,
    float alpha,
    float const *A,
    int lda,
    float const *B,
    int ldb,
    float beta,
    float *C,
    int ldc)
{


    // using ColumnMajor = cutlass::layout::ColumnMajor;
    using RowMajor = cutlass::layout::RowMajor;

    using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                    RowMajor,  // Layout of A matrix
                                                    float,        // Data-type of B matrix
                                                    RowMajor,  // Layout of B matrix
                                                    float,        // Data-type of C matrix
                                                    RowMajor>; // Layout of C matrix

    // Define a CUTLASS GEMM type
    CutlassGemm gemm_operator;

    CutlassGemm::Arguments args({M, N, K},      // Gemm Problem dimensions
                                {A, lda},       // Tensor-ref for source matrix A
                                {B, ldb},       // Tensor-ref for source matrix B
                                {C, ldc},       // Tensor-ref for source matrix C
                                {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                                {alpha, beta}); // Scalars used in the Epilogue

    cutlass::Status status = gemm_operator(args);

    if (status != cutlass::Status::kSuccess)
    {
        return hipErrorUnknown;
    }

    // Return success, if no errors were encountered.
    return hipSuccess;
}



/// Allocate several matrices in GPU device memory and call a single-precision
/// CUTLASS GEMM kernel.
void launcher_CutlassGemmNN_float(const float* mat_A, const float* mat_B, float* mat_C, int M, int N, int K, float alpha, float beta)
{
    // Compute leading dimensions for each matrix. 
    // 如果以行主序的话，则leading dim应该是列维度值
    int lda = K;
    int ldb = N;
    int ldc = N;

    CutlassSgemmNN_float(M, N, K, alpha, mat_A, lda, mat_B, ldb, beta, mat_C, ldc);
}
